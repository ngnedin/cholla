
#include <hip/hip_runtime.h>
#ifdef GRAVITY
#ifdef CUFFT

#include "../gravity/potential_CUFFT_3D.h"
#include "../global/global_cuda.h"
#include "../io/io.h"
#include <iostream>


Potential_CUFFT_3D::Potential_CUFFT_3D( void ){}

void Potential_CUFFT_3D::Initialize( Real Lx, Real Ly, Real Lz, Real x_min, Real y_min, Real z_min, 
                                     int nx, int ny, int nz, int nx_real, int ny_real, int nz_real, 
                                     Real dx_real, Real dy_real, Real dz_real)
{
  //
  Lbox_x = Lx;
  Lbox_y = Ly;
  Lbox_z = Lz;

  nx_total = nx;
  ny_total = ny;
  nz_total = nz;

  nx_local = nx_real;
  ny_local = ny_real;
  nz_local = nz_real;

  dx = dx_real;
  dy = dy_real;
  dz = dz_real;

  n_cells_local = nx_local*ny_local*nz_local;
  n_cells_total = nx_total*ny_total*nz_total;
  chprintf( " Using Poisson Solver: CUFFT\n");
  chprintf( "  CUFFT: L[ %f %f %f ] N[ %d %d %d ] dx[ %f %f %f ]\n", Lbox_x, Lbox_y, Lbox_z, nx_local, ny_local, nz_local, dx, dy, dz );

  chprintf( "  CUFFT: Allocating memory...\n");
  AllocateMemory_CPU();
  AllocateMemory_GPU();

  chprintf( "  CUFFT: Creating FFT plan...\n");
  cufftPlan3d( &plan_cufft_fwd,  nz_local, ny_local,  nx_local, CUFFT_Z2Z);
  cufftPlan3d( &plan_cufft_bwd,  nz_local, ny_local,  nx_local, CUFFT_Z2Z);

  chprintf( "  CUFFT: Computing K for Gravity Green Function\n");
  cudaMalloc( (void**)&F.G_d, n_cells_local*sizeof(Real));
  Get_K_for_Green_function();
  threads_per_block = 512;
  blocks_per_grid = (( n_cells_local - 1 ) / threads_per_block) + 1;
  chprintf( "  CUFFT: Using %d threads and %d blocks for applying G funtion: %d \n", threads_per_block, blocks_per_grid, threads_per_block*blocks_per_grid);

}

void Potential_CUFFT_3D::AllocateMemory_CPU( void )
{
  F.output_h = (Complex_cufft *) malloc(n_cells_local*sizeof(Complex_cufft));
  F.G_h = (Real *) malloc(n_cells_local*sizeof(Real_cufft));
}

void Potential_CUFFT_3D::AllocateMemory_GPU( void )
{
  cudaMalloc( (void**)&F.input_real_d, n_cells_local*sizeof(Real_cufft));
  cudaMalloc( (void**)&F.input_d, n_cells_local*sizeof(Complex_cufft));
  cudaMalloc( (void**)&F.transform_d, n_cells_local*sizeof(Complex_cufft));
  cudaMalloc( (void**)&F.output_d, n_cells_local*sizeof(Complex_cufft));
  cudaMalloc( (void**)&F.G_d, n_cells_local*sizeof(Real_cufft));
  CudaCheckError();
}

void Potential_CUFFT_3D::FreeMemory_GPU( void )
{
  cudaFree( F.input_real_d );
  cudaFree( F.input_d );
  cudaFree( F.output_d );
  cudaFree( F.transform_d );
  cudaFree( F.G_d );
  CudaCheckError();
}

void Potential_CUFFT_3D::Reset( void )
{
  // chprintf("Reset CUFFT\n");
  free( F.output_h );
  free( F.G_h );
  FreeMemory_GPU();
}


void Potential_CUFFT_3D::Get_K_for_Green_function( void)
{
  Real kx, ky, kz, Gx, Gy, Gz, G;
  int id;
  for (int k=0; k<nz_local; k++){
    kz =  2*M_PI*k/nz_local;
    Gz = sin( kz/2 );
    for (int j=0; j<ny_local; j++){
      ky =  2*M_PI*j/ny_local;
      Gy = sin( ky/2 );
      for ( int i=0; i<nx_local; i++){
        id = i + j*nx_local + k*nx_local*ny_local;
        kx =  2*M_PI*i/nx_local;
        Gx = sin( kx/2 );
        G = -1 / ( Gx*Gx + Gy*Gy + Gz*Gz ) * dx * dx / 4 ;
        if ( id == 0 ) G = 1;
        F.G_h[id] = G;
        // F.G_h[id] = 0.1;
      }
    }
  }
  cudaMemcpy( F.G_d, F.G_h, n_cells_local*sizeof(Real), cudaMemcpyHostToDevice );
  CudaCheckError();
}

__global__
void Copy_Input_Kernel( int n_cells, Real *input_h, Complex_cufft *input_d, Real Grav_Constant, Real dens_avrg, Real current_a )
{
  int t_id = threadIdx.x + blockIdx.x*blockDim.x;
  if ( t_id < n_cells ){
    #ifdef COSMOLOGY
    input_d[t_id].x = 4 * M_PI * Grav_Constant * ( input_h[t_id] - dens_avrg ) / current_a;
    #else
    input_d[t_id].x = 4 * M_PI * Grav_Constant * input_h[t_id];
    #endif
    input_d[t_id].y = 0.0;
  }
}

void Potential_CUFFT_3D::Copy_Input( Real *input_density, Real Grav_Constant, Real dens_avrg, Real current_a )
{
  cudaMemcpy( F.input_real_d, input_density, n_cells_local*sizeof(Real_cufft), cudaMemcpyHostToDevice );
  hipLaunchKernelGGL(  Copy_Input_Kernel, blocks_per_grid,  threads_per_block, 0, 0,  n_cells_local, 
                       F.input_real_d, F.input_d, Grav_Constant, dens_avrg, current_a );
}

void Potential_CUFFT_3D::Copy_Output( Real *output_potential )
{

  cudaMemcpy( F.output_h, F.output_d, n_cells_local*sizeof(Complex_cufft), cudaMemcpyDeviceToHost );

  int id, id_pot;
  int i, k, j;
  for (k=0; k<nz_local; k++) {
    for (j=0; j<ny_local; j++) {
      for (i=0; i<nx_local; i++) {
        id = i + j*nx_local + k*nx_local*ny_local;
        id_pot = (i+N_GHOST_POTENTIAL) + (j+N_GHOST_POTENTIAL)*(nx_local+2*N_GHOST_POTENTIAL) + (k+N_GHOST_POTENTIAL)*(nx_local+2*N_GHOST_POTENTIAL)*(ny_local+2*N_GHOST_POTENTIAL);
        output_potential[id_pot] = F.output_h[id].x / n_cells_local;
      }
    }
  }
}



__global__
void Apply_G_Function( int n_cells, Complex_cufft *transform, Real *G )
{
  int t_id = threadIdx.x + blockIdx.x*blockDim.x;
  Real G_val;
  if ( t_id < n_cells ){
    G_val = G[t_id];
    if ( t_id == 0 ) G_val = 1.0;
    transform[t_id].x *= G_val;
    transform[t_id].y *= G_val;
    if ( t_id == 0 ){
      transform[t_id].x = 0;
      transform[t_id].y = 0;
    }
  }
}


Real Potential_CUFFT_3D::Get_Potential( Real *input_density,  Real *output_potential, Real Grav_Constant, Real dens_avrg, Real current_a )
{
  //
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  cudaEventRecord(start);
  //
  // AllocateMemory_GPU();
  Copy_Input( input_density, Grav_Constant, dens_avrg, current_a );

  cufftExecZ2Z( plan_cufft_fwd, F.input_d, F.transform_d, CUFFT_FORWARD );
hipLaunchKernelGGL(  Apply_G_Function, blocks_per_grid,  threads_per_block, 0, 0,  n_cells_local, F.transform_d, F.G_d );
  cufftExecZ2Z( plan_cufft_bwd, F.transform_d, F.output_d, CUFFT_INVERSE );

  Copy_Output( output_potential );
  //
  // FreeMemory_GPU();
  //
  cudaEventRecord(stop);
  cudaEventSynchronize(stop);
  float milliseconds = 0;
  cudaEventElapsedTime(&milliseconds, start, stop);
  // chprintf( " CUFFT: Potential Time = %f   msecs\n", milliseconds);
  // return (Real) milliseconds;

  return 0;

}

#endif //POTENTIAL_CUFFT
#endif //GRAVITY
