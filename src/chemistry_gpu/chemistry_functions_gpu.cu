
#include <hip/hip_runtime.h>
#ifdef CHEMISTRY_GPU

  #include "../global/global_cuda.h"
  #include "../hydro/hydro_cuda.h"
  #include "../io/io.h"
  #include "../radiation/alt/static_table_gpu.cuh"
  #include "chemistry_gpu.h"
  #include "rates.cuh"
  #include "rates_Katz95.cuh"

//  #define eV_to_K 1.160451812e4
//  #define K_to_eV 8.617333263e-5
//  #define n_min   1e-20
//  #define TINY_NUMBER    1e-20

  #define TPB_CHEM 256

//Allocate a floating point array on the gpu
void Chem_GPU::Allocate_Array_GPU_float(float **array_dev, int size)
{
  CudaSafeCall(cudaMalloc((void **)array_dev, size * sizeof(float)));
}

//Copy a floating point array from the cpu to the gpu
void Chem_GPU::Copy_Float_Array_to_Device(int size, float *array_h, float *array_d)
{
  CudaSafeCall(cudaMemcpy(array_d, array_h, size * sizeof(float), cudaMemcpyHostToDevice));
  cudaDeviceSynchronize();
}

//Free an array allocated on the gpu
void Chem_GPU::Free_Array_GPU_float(float *array_dev) { CudaSafeCall(cudaFree(array_dev)); }


//Allocate a Real array on the gpu
void Chem_GPU::Allocate_Array_GPU_Real(Real **array_dev, int size)
{
  CudaSafeCall(cudaMalloc((void **)array_dev, size * sizeof(Real)));
}

//Copy a Real array from the cpu to the gpu
void Chem_GPU::Copy_Real_Array_to_Device(int size, Real *array_h, Real *array_d)
{
  CudaSafeCall(cudaMemcpy(array_d, array_h, size * sizeof(Real), cudaMemcpyHostToDevice));
  cudaDeviceSynchronize();
}

//Free a Real array on the gpu
void Chem_GPU::Free_Array_GPU_Real(Real *array_dev) { CudaSafeCall(cudaFree(array_dev)); }


//compute the thermal state
//of a cell based on its ionization state
class Thermal_State
{
 public:
  Real U;         //internal energy in (km/s)^2
  Real d;         //total density
  Real d_HI;      //neutral hydrogen density
  Real d_HII;     //ionized hydrogen density
  Real d_HeI;     //neutral helium density
  Real d_HeII;    //singly ionized helium density
  Real d_HeIII;   //doubly ionized helium density
  Real d_e;       //electron density

  // Constructor
  __host__ __device__ Thermal_State(Real U_0 = 1, Real d_0 = 1, Real d_HI_0 = 1, Real d_HII_0 = 0, Real d_HeI_0 = 1,
                                    Real d_HeII_0 = 0, Real d_HeIII_0 = 1, Real d_e_0 = 0)
      : U(U_0), d(d_0), d_HI(d_HI_0), d_HII(d_HII_0), d_HeI(d_HeI_0), d_HeII(d_HeII_0), d_HeIII(d_HeIII_0), d_e(d_e_0)
  {
  }

  //Compute the mean molecular weight
  __host__ __device__ Real get_MMW()
  {
    // Real m_tot = d_HI + d_HII + d_HeI + d_HeII + d_HeIII;
    Real n_tot = d_HI + d_HII + 0.25 * (d_HeI + d_HeII + d_HeIII) + d_e;
    return d / n_tot;
    // return m_tot / n_tot;
  }

  //Compute the temperature from the internal energy
  //mean molecular weight, proton mass, Boltzmann's constant,
  //the adiabatic index, and converting from KM to CGS
  //returns the temperature in Kelvin
  __host__ __device__ Real get_temperature(Real gamma)
  {
    Real mu, temp;
    mu   = get_MMW();
    temp = (gamma - 1) * mu * U * MP * (KM_CGS*KM_CGS) / KB ;
    return temp;
  }

  //Compute the internal energy from the temperature in K,
  //the adiabatic index, the mean molecular weight, the
  //mass of the proton, Boltzmann's constant, and the 
  //conversion from CGS to KM, returns the
  //internal energy in (km/s)^2
  __host__ __device__ Real compute_U(Real temp, Real gamma)
  {
    Real mu, U_local;
    mu      = get_MMW();
    U_local = KB * temp / (gamma - 1) / mu / MP / (KM_CGS*KM_CGS);
    return U_local;
  }
};

//get the integer index and fractional index in the look up table
//for a temperature T
__device__ void get_temperature_indx(Real T, Chemistry_Header &Chem_H, int &temp_indx, Real &delta_T, Real temp_old,
                                     int print)
{
  Real logT, logT_start, d_logT, logT_l, logT_r;
  logT       = log(0.5 * (T + temp_old));
  logT_start = log(Chem_H.Temp_start);
  logT       = fmax(logT_start, logT);
  logT       = fmin(log(Chem_H.Temp_end), logT);
  d_logT     = (log(Chem_H.Temp_end) - logT_start) / (Chem_H.N_Temp_bins - 1);
  temp_indx  = (int)floor((logT - logT_start) / d_logT);
  temp_indx  = max(0, temp_indx);
  temp_indx  = min(Chem_H.N_Temp_bins - 2, temp_indx);
  logT_l     = logT_start + temp_indx * d_logT;
  logT_r     = logT_start + (temp_indx + 1) * d_logT;
  delta_T    = (logT - logT_l) / (logT_r - logT_l);
  // if (print) printf(" logT_start: %f  logT_end: %f  d_logT: %f   \n", logT_start, log( Chem_H.Temp_end ), d_logT );
  // if (print) printf(" logT: %f  logT_l: %f  logT_r: %f   \n", logT, logT_l, logT_r );
}


//perform a linear interpolation on a rate table given the integer
//and fractional index
__device__ Real interpolate_rate(Real *rate_table, int indx, Real delta)
{
  Real rate_val;
  rate_val = rate_table[indx];
  rate_val = rate_val + delta * (rate_table[indx + 1] - rate_val);
  return rate_val;
}

__device__ Real Get_Cooling_Rates(Thermal_State &TS, Chemistry_Header &Chem_H, Real dens_number_conv, Real current_z,
                                  Real temp_prev, float photo_h_HI, float photo_h_HeI, float photo_h_HeII, int print)
{
  int temp_indx;
  Real temp, delta_T, U_dot;
  temp = TS.get_temperature(Chem_H.gamma);
  get_temperature_indx(temp, Chem_H, temp_indx, delta_T, temp_prev, print);
  if (print > 1) printf("mu: %f  temp: %f  temp_indx: %d  delta_T: %f  \n", TS.get_MMW(), temp, temp_indx, delta_T);
  U_dot = 0.0;

  // Collisional excitation cooling
  Real cool_ceHI, cool_ceHeI, cool_ceHeII;
  cool_ceHI = interpolate_rate(Chem_H.cool_ceHI_d, temp_indx, delta_T) * TS.d_HI * TS.d_e;
  cool_ceHeI =
      interpolate_rate(Chem_H.cool_ceHeI_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e * TS.d_e * dens_number_conv / 4.0;
  cool_ceHeII = interpolate_rate(Chem_H.cool_ceHeII_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  U_dot -= cool_ceHI + cool_ceHeI + cool_ceHeII;

  // Collisional excitation cooling
  Real cool_ciHI, cool_ciHeI, cool_ciHeII, cool_ciHeIS;
  cool_ciHI   = interpolate_rate(Chem_H.cool_ciHI_d, temp_indx, delta_T) * TS.d_HI * TS.d_e;
  cool_ciHeI  = interpolate_rate(Chem_H.cool_ciHeI_d, temp_indx, delta_T) * TS.d_HeI * TS.d_e / 4.0;
  cool_ciHeII = interpolate_rate(Chem_H.cool_ciHeII_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  cool_ciHeIS =
      interpolate_rate(Chem_H.cool_ciHeIS_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e * TS.d_e * dens_number_conv / 4.0;
  U_dot -= cool_ciHI + cool_ciHeI + cool_ciHeII + cool_ciHeIS;

  // Recombination cooling
  Real cool_reHII, cool_reHeII1, cool_reHeII2, cool_reHeIII;
  cool_reHII   = interpolate_rate(Chem_H.cool_reHII_d, temp_indx, delta_T) * TS.d_HII * TS.d_e;
  cool_reHeII1 = interpolate_rate(Chem_H.cool_reHeII1_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  cool_reHeII2 = interpolate_rate(Chem_H.cool_reHeII2_d, temp_indx, delta_T) * TS.d_HeII * TS.d_e / 4.0;
  cool_reHeIII = interpolate_rate(Chem_H.cool_reHeIII_d, temp_indx, delta_T) * TS.d_HeIII * TS.d_e / 4.0;
  U_dot -= cool_reHII + cool_reHeII1 + cool_reHeII2 + cool_reHeIII;

  // Bremsstrahlung cooling
  Real cool_brem;
  cool_brem =
      interpolate_rate(Chem_H.cool_brem_d, temp_indx, delta_T) * (TS.d_HII + TS.d_HeII / 4.0 + TS.d_HeIII) * TS.d_e;
  U_dot -= cool_brem;

  #ifdef COSMOLOGY
  // Compton cooling or heating
  Real cool_compton, temp_cmb;
  temp_cmb     = 2.73 * (1.0 + current_z);
  cool_compton = Chem_H.cool_compton * pow(1.0 + current_z, 4) * (temp - temp_cmb) * TS.d_e / dens_number_conv;
  U_dot -= cool_compton;
  #endif

  // Phothoheating
  Real photo_heat;
  photo_heat = (photo_h_HI * TS.d_HI + 0.25 * (photo_h_HeI * TS.d_HeI + photo_h_HeII * TS.d_HeII)) / dens_number_conv;
  U_dot += photo_heat;

  if (temp <= 1.01 * Chem_H.Temp_start && fabs(U_dot) < 0) U_dot = TINY_NUMBER;
  if (fabs(U_dot) < TINY_NUMBER) U_dot = TINY_NUMBER;

  if (print > 1) printf("HI: %e  \n", TS.d_HI);
  if (print > 1) printf("HII: %e  \n", TS.d_HII);
  if (print > 1) printf("HeI: %e  \n", TS.d_HeI);
  if (print > 1) printf("HeII: %e  \n", TS.d_HeII);
  if (print > 1) printf("HeIII: %e  \n", TS.d_HeIII);
  if (print > 1) printf("de: %e  \n", TS.d_e);
  if (print > 1) printf("Cooling ceHI: %e  \n", cool_ceHI);
  if (print > 1) printf("Cooling ceHeI: %e   \n", cool_ceHeI);
  if (print > 1) printf("Cooling ceHeII: %e   \n", cool_ceHeII);
  if (print > 1) printf("Cooling ciHI: %e  \n", cool_ciHI);
  if (print > 1) printf("Cooling ciHeI: %e  \n", cool_ciHeI);
  if (print > 1) printf("Cooling ciHeII: %e  \n", cool_ciHeII);
  if (print > 1) printf("Cooling ciHeIS: %e  \n", cool_ciHeIS);
  if (print > 1) printf("Cooling reHII: %e  \n", cool_reHII);
  if (print > 1) printf("Cooling reHeII1: %e  \n", cool_reHeII1);
  if (print > 1) printf("Cooling reHeII2: %e  \n", cool_reHeII2);
  if (print > 1) printf("Cooling reHeIII: %e  \n", cool_reHeIII);
  if (print > 1) printf("Cooling brem: %e  \n", cool_brem);
  if (print > 0) printf("Cooling piHI: %e   rate: %e \n", photo_h_HI, photo_h_HI * TS.d_HI / dens_number_conv);
  if (print > 1)
    printf("Cooling piHeI: %e  rate: %e \n", photo_h_HeI, photo_h_HeI * TS.d_HeI / dens_number_conv * 0.25);
  if (print > 1)
    printf("Cooling piHeII: %e rate: %e \n", photo_h_HeII, photo_h_HeII * TS.d_HeII / dens_number_conv * 0.25);
  if (print > 1) printf("Cooling DOM: %e  \n", dens_number_conv);
  #ifdef COSMOLOGY
  if (print > 1) printf("Cooling compton: %e  \n", cool_compton);
  #endif
  if (print > 0) printf("Cooling U_dot: %e  \n", U_dot);

  return U_dot;
}

__device__ void Get_Reaction_Rates(Thermal_State &TS, Chemistry_Header &Chem_H, Real &k_coll_i_HI, Real &k_coll_i_HeI,
                                   Real &k_coll_i_HeII, Real &k_coll_i_HI_HI, Real &k_coll_i_HI_HeI, Real &k_recomb_HII,
                                   Real &k_recomb_HeII, Real &k_recomb_HeIII, int print)
{
  int temp_indx;
  Real temp, delta_T;
  temp = TS.get_temperature(Chem_H.gamma);
  get_temperature_indx(temp, Chem_H, temp_indx, delta_T, temp, print);

  k_coll_i_HI   = interpolate_rate(Chem_H.k_coll_i_HI_d, temp_indx, delta_T);
  k_coll_i_HeI  = interpolate_rate(Chem_H.k_coll_i_HeI_d, temp_indx, delta_T);
  k_coll_i_HeII = interpolate_rate(Chem_H.k_coll_i_HeII_d, temp_indx, delta_T);

  k_coll_i_HI_HI  = interpolate_rate(Chem_H.k_coll_i_HI_HI_d, temp_indx, delta_T);
  k_coll_i_HI_HeI = interpolate_rate(Chem_H.k_coll_i_HI_HeI_d, temp_indx, delta_T);

  k_recomb_HII   = interpolate_rate(Chem_H.k_recomb_HII_d, temp_indx, delta_T);
  k_recomb_HeII  = interpolate_rate(Chem_H.k_recomb_HeII_d, temp_indx, delta_T);
  k_recomb_HeIII = interpolate_rate(Chem_H.k_recomb_HeIII_d, temp_indx, delta_T);

  if (print > 1) printf("logT: %f   temp_indx: %d\n", log(temp), temp_indx);
  if (print > 1) printf("k_coll_i_HI: %e \n", k_coll_i_HI);
  if (print > 1) printf("k_coll_i_HeI: %e \n", k_coll_i_HeI);
  if (print > 1) printf("k_coll_i_HeII: %e \n", k_coll_i_HeII);
  if (print > 1) printf("k_coll_i_HI_HI: %e \n", k_coll_i_HI_HI);
  if (print > 1) printf("k_coll_i_HI_HeI: %e \n", k_coll_i_HI_HeI);
  if (print > 1) printf("k_recomb_HII: %e \n", k_recomb_HII);
  if (print > 1) printf("k_recomb_HeII: %e \n", k_recomb_HeII);
  if (print > 1) printf("k_recomb_HeIII: %e \n", k_recomb_HeIII);
}

__device__ int Binary_Search(int N, Real val, float *data, int indx_l, int indx_r)
{
  int n, indx;
  n    = indx_r - indx_l;
  indx = indx_l + n / 2;
  if (val >= data[N - 1]) return indx_r;
  if (val <= data[0]) return indx_l;
  if (indx_r == indx_l + 1) return indx_l;
  if (data[indx] <= val)
    indx_l = indx;
  else
    indx_r = indx;
  return Binary_Search(N, val, data, indx_l, indx_r);
}


//Another function to perform linear interpolation on a float array
__device__ Real linear_interpolation(Real delta_x, int indx_l, int indx_r, float *array)
{
  float v_l, v_r;
  Real v;
  v_l = array[indx_l];
  v_r = array[indx_r];
  v   = delta_x * (v_r - v_l) + v_l;
  return v;
}

__device__ void Get_Current_Photo_Rates(Chemistry_Header &Chem_H, const Real *rf, int id, int ncells, float &photo_i_HI,
                                        float &photo_i_HeI, float &photo_i_HeII, float &photo_h_HI, float &photo_h_HeI,
                                        float &photo_h_HeII, int print)
{
  #ifdef RT
  if (rf != nullptr) {
    const float rfN0    = rf[id + 0 * ncells];
    const float rfNHI   = rf[id + 1 * ncells];
    const float rfNHeI  = rf[id + 2 * ncells];
    const float rfNHeII = rf[id + 3 * ncells];

    float tauHI   = (rfNHI > rfN0 ? 0 : (rfNHI > 0 ? -log(1.0e-35 + rfNHI / rfN0) : 1001));
    float tauHeI  = (rfNHeI > rfN0 ? 0 : (rfNHeI > 0 ? -log(1.0e-35 + rfNHeI / rfN0) : 1001));
    float tauHeII = (rfNHeII > rfN0 ? 0 : (rfNHeII > 0 ? -log(1.0e-35 + rfNHeII / rfN0) : 1001));

    float pRates[6];
    float x[3] = {Chem_H.dStretch->tau2x(tauHI), Chem_H.dStretch->tau2x(tauHeI), Chem_H.dStretch->tau2x(tauHeII)};
    Chem_H.dTables[0]->GetValues(x, pRates, 0, 6);

    for (unsigned int i = 0; i < 6; i++) {
      pRates[i] *= rfN0;
    }

    if (Chem_H.dTables[1] != nullptr) {
      const float rfFHI   = rf[id + 4 * ncells];
      const float rfFHeI  = rf[id + 5 * ncells];
      const float rfFHeII = rf[id + 6 * ncells];

      float tauHI   = (rfFHI > 1 ? 0 : (rfFHI > 0 ? -log(1.0e-35 + rfFHI) : 1001));
      float tauHeI  = (rfFHeI > 1 ? 0 : (rfFHeI > 0 ? -log(1.0e-35 + rfFHeI) : 1001));
      float tauHeII = (rfFHeII > 1 ? 0 : (rfFHeII > 0 ? -log(1.0e-35 + rfFHeII) : 1001));

      x[0] = Chem_H.dStretch->tau2x(tauHI);
      x[1] = Chem_H.dStretch->tau2x(tauHeI);
      x[2] = Chem_H.dStretch->tau2x(tauHeII);

      float pRates2[6];
      Chem_H.dTables[1]->GetValues(x, pRates2, 0, 6);

      for (unsigned int i = 0; i < 6; i++) {
        pRates[i] += pRates2[i];
      }
    }

    photo_i_HI   = pRates[0] * Chem_H.unitPhotoIonization;
    photo_h_HI   = pRates[1] * Chem_H.unitPhotoHeating;
    photo_i_HeI  = pRates[2] * Chem_H.unitPhotoIonization;
    photo_h_HeI  = pRates[3] * Chem_H.unitPhotoHeating;
    photo_i_HeII = pRates[4] * Chem_H.unitPhotoIonization;
    photo_h_HeII = pRates[5] * Chem_H.unitPhotoHeating;
  } else
  #endif
  {
    if (Chem_H.current_z > Chem_H.uvb_rates_redshift_d[Chem_H.n_uvb_rates_samples - 1]) {
      photo_h_HI   = 0;
      photo_h_HeI  = 0;
      photo_h_HeII = 0;
      photo_i_HI   = 0;
      photo_i_HeI  = 0;
      photo_i_HeII = 0;
      return;
    }

    // Find closest value of z in rates_z such that z<=current_z
    int indx_l;
    Real z_l, z_r, delta_x;
    indx_l  = Binary_Search(Chem_H.n_uvb_rates_samples, Chem_H.current_z, Chem_H.uvb_rates_redshift_d, 0,
                            Chem_H.n_uvb_rates_samples - 1);
    z_l     = Chem_H.uvb_rates_redshift_d[indx_l];
    z_r     = Chem_H.uvb_rates_redshift_d[indx_l + 1];
    delta_x = (Chem_H.current_z - z_l) / (z_r - z_l);

    photo_i_HI   = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_ion_HI_rate_d);
    photo_i_HeI  = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_ion_HeI_rate_d);
    photo_i_HeII = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_ion_HeII_rate_d);
    photo_h_HI   = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_heat_HI_rate_d);
    photo_h_HeI  = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_heat_HeI_rate_d);
    photo_h_HeII = linear_interpolation(delta_x, indx_l, indx_l + 1, Chem_H.photo_heat_HeII_rate_d);
  }
}



// Determine the chemistry timestep
__device__ Real Get_Chemistry_dt(Thermal_State &TS, Chemistry_Header &Chem_H, Real &HI_dot, Real &e_dot, Real U_dot,
                                 Real k_coll_i_HI, Real k_coll_i_HeI, Real k_coll_i_HeII, Real k_coll_i_HI_HI,
                                 Real k_coll_i_HI_HeI, Real k_recomb_HII, Real k_recomb_HeII, Real k_recomb_HeIII,
                                 float photo_i_HI, float photo_i_HeI, float photo_i_HeII, int n_iter, Real HI_dot_prev,
                                 Real e_dot_prev, Real t_chem, Real dt_hydro, int print)
{
  Real dt, energy;
  // Rate of change of HI
  HI_dot = k_recomb_HII * TS.d_HII * TS.d_e - k_coll_i_HI * TS.d_HI * TS.d_e - k_coll_i_HI_HI * TS.d_HI * TS.d_HI -
           k_coll_i_HI_HeI * TS.d_HI * TS.d_HeI / 4.0 - photo_i_HI * TS.d_HI;

  // Rate of change of electron
  e_dot = k_coll_i_HI * TS.d_HI * TS.d_e + k_coll_i_HeI * TS.d_HeI / 4.0 * TS.d_e +
          k_coll_i_HeII * TS.d_HeII / 4.0 * TS.d_e + k_coll_i_HI_HI * TS.d_HI * TS.d_HI +
          +k_coll_i_HI_HeI * TS.d_HI * TS.d_HeI / 4.0 - k_recomb_HII * TS.d_HII * TS.d_e -
          k_recomb_HeII * TS.d_HeII / 4.0 * TS.d_e - k_recomb_HeIII * TS.d_HeIII / 4.0 * TS.d_e + photo_i_HI * TS.d_HI +
          photo_i_HeI * TS.d_HeI / 4.0 + photo_i_HeII * TS.d_HeII / 4.0;

  // Bound from below to prevent numerical errors
  if (fabs(HI_dot) < TINY_NUMBER) HI_dot = fmin(TINY_NUMBER, TS.d_HI);
  if (fabs(e_dot) < TINY_NUMBER) e_dot = fmin(TINY_NUMBER, TS.d_e);

  // If the net rate is almost perfectly balanced then set
  // it to zero (since it is zero to available precision)
  if (fmin(fabs(k_coll_i_HI * TS.d_HI * TS.d_e), fabs(k_recomb_HII * TS.d_HII * TS.d_e)) /
          fmax(fabs(HI_dot), fabs(e_dot)) >
      1e6) {
    HI_dot = TINY_NUMBER;
    e_dot  = TINY_NUMBER;
  }

  if (n_iter > 50) {
    HI_dot = fmin(fabs(HI_dot), fabs(HI_dot_prev));
    e_dot  = fmin(fabs(e_dot), fabs(e_dot_prev));
  }

  if (TS.d * Chem_H.dens_number_conv > 1e8 && U_dot > 0) {
    printf("#### Equlibrium  \n");
  }

  #ifdef TEMPERATURE_FLOOR
  if (TS.get_temperature(Chem_H.gamma) < TEMP_FLOOR) TS.U = TS.compute_U(TEMP_FLOOR, Chem_H.gamma);
  #endif

  energy = fmax(TS.U * TS.d, TINY_NUMBER);

  //limit the timestep
  dt = fabs(0.1 * TS.d_HI / HI_dot);
  dt = fmin(fabs(0.1 * energy / U_dot), dt);
  dt = fmin(0.5 * dt_hydro, dt);
  dt = fmin(dt_hydro - t_chem, dt);

  if (n_iter == Chem_H.max_iter - 1) {
    // printf("##### Chem_GPU: dt_hydro: %e   t_chem: %e   dens: %e   temp: %e  GE: %e  U_dot: %e   dt_HI: %e   dt_e: %e
    // dt_U: %e \n", dt_hydro,  t_chem, TS.d, TS.get_temperature(Chem_H.gamma), energy, U_dot, fabs( 0.1 * TS.d_HI /
    // HI_dot ), fabs( 0.1 * TS.d_e / e_dot ), fabs( 0.1 * TS.U * TS.d / U_dot )   ) ;
  }

  if (print > 0) {
    printf("HIdot: %e (%g,%g)\n", HI_dot, k_recomb_HII * TS.d_HII * TS.d_e, photo_i_HI * TS.d_HI);
  }
  if (print > 1) printf("edot: %e\n", e_dot);
  if (print > 1) printf("energy: %e\n", TS.U * TS.d);
  if (print > 1) printf("Udot: %e\n", U_dot);
  if (print > 1) printf("dt_hydro: %e\n", dt_hydro);
  if (print > 1) printf("dt: %e\n", dt);

  //return the chemistry timestep
  return dt;
}


// update the ionization fractions 
// and internal energy
__device__ void Update_Step(Thermal_State &TS, Chemistry_Header &Chem_H, Real dt, Real U_dot, Real k_coll_i_HI,
                            Real k_coll_i_HeI, Real k_coll_i_HeII, Real k_coll_i_HI_HI, Real k_coll_i_HI_HeI,
                            Real k_recomb_HII, Real k_recomb_HeII, Real k_recomb_HeIII, float photo_i_HI,
                            float photo_i_HeI, float photo_i_HeII, Real &HI_dot_prev, Real &e_dot_prev, Real &temp_prev,
                            int print)
{
  Real d_HI_p, d_HII_p, d_HeI_p, d_HeII_p, d_HeIII_p, d_e_p;
  Real s_coef, a_coef;

  // Update HI
  s_coef = k_recomb_HII * TS.d_HII * TS.d_e;
  a_coef = k_coll_i_HI * TS.d_e + k_coll_i_HI_HI * TS.d_HI + k_coll_i_HI_HeI * TS.d_HeI / 4.0 + photo_i_HI;
  d_HI_p = (dt * s_coef + TS.d_HI) / (1.0 + dt * a_coef);
  if (print > 1) printf("Update HI  s_coef: %e    a_coef: %e   HIp: %e \n", s_coef, a_coef, d_HI_p);

  // Update HII
  s_coef = k_coll_i_HI * d_HI_p * TS.d_e + k_coll_i_HI_HI * d_HI_p * d_HI_p +
           k_coll_i_HI_HeI * d_HI_p * TS.d_HeI / 4.0 + photo_i_HI * d_HI_p;
  a_coef  = k_recomb_HII * TS.d_e;
  d_HII_p = (dt * s_coef + TS.d_HII) / (1.0 + dt * a_coef);
  if (print > 1) printf("Update HII  s_coef: %e    a_coef: %e   HIIp: %e \n", s_coef, a_coef, d_HII_p);

  // Update electron
  s_coef = k_coll_i_HI_HI * d_HI_p * d_HI_p + k_coll_i_HI_HeI * d_HI_p * TS.d_HeI / 4.0 + photo_i_HI * TS.d_HI +
           photo_i_HeI * TS.d_HeI / 4.0 + photo_i_HeII * TS.d_HeII / 4.0;
  a_coef = -k_coll_i_HI * TS.d_HI + k_recomb_HII * TS.d_HII - k_coll_i_HeI * TS.d_HeI / 4.0 +
           k_recomb_HeII * TS.d_HeII / 4.0 - k_coll_i_HeII * TS.d_HeII / 4.0 + k_recomb_HeIII * TS.d_HeIII / 4.0;
  d_e_p = (dt * s_coef + TS.d_e) / (1.0 + dt * a_coef);
  if (print > 1) printf("Update e  s_coef: %e    a_coef: %e   ep: %e \n", s_coef, a_coef, d_e_p);

  // Update HeI
  s_coef  = k_recomb_HeII * TS.d_HeII * TS.d_e;
  a_coef  = k_coll_i_HeI * TS.d_e + photo_i_HeI;
  d_HeI_p = (dt * s_coef + TS.d_HeI) / (1.0 + dt * a_coef);
  if (print > 1) printf("Update HeI  s_coef: %e    a_coef: %e   HeIp: %e \n", s_coef, a_coef, d_HeI_p);

  // Update HeII
  s_coef   = k_coll_i_HeI * d_HeI_p * TS.d_e + k_recomb_HeIII * TS.d_HeIII * TS.d_e + photo_i_HeI * d_HeI_p;
  a_coef   = k_recomb_HeII * TS.d_e + k_coll_i_HeII * TS.d_e + photo_i_HeII;
  d_HeII_p = (dt * s_coef + TS.d_HeII) / (1.0 + dt * a_coef);
  if (print > 1) printf("Update HeII  s_coef: %e    a_coef: %e   HeIIp: %e \n", s_coef, a_coef, d_HeII_p);

  // Update HeIII
  s_coef    = k_coll_i_HeII * d_HeII_p * TS.d_e + photo_i_HeII * d_HeII_p;
  a_coef    = k_recomb_HeIII * TS.d_e;
  d_HeIII_p = (dt * s_coef + TS.d_HeIII) / (1.0 + dt * a_coef);
  if (print > 1) printf("Update HeIII  s_coef: %e    a_coef: %e   HeIIIp: %e \n", s_coef, a_coef, d_HeIII_p);

  // Record the temperature for the next step
  temp_prev = TS.get_temperature(Chem_H.gamma);

  HI_dot_prev = fabs(TS.d_HI - d_HI_p) / fmax(dt, TINY_NUMBER);
  TS.d_HI     = fmax(d_HI_p, TINY_NUMBER);
  TS.d_HII    = fmax(d_HII_p, TINY_NUMBER);
  TS.d_HeI    = fmax(d_HeI_p, TINY_NUMBER);
  TS.d_HeII   = fmax(d_HeII_p, TINY_NUMBER);
  TS.d_HeIII  = fmax(d_HeIII_p, 1e-5 * TINY_NUMBER);

  // Use charge conservation to determine electron fraction
  e_dot_prev = TS.d_e;
  TS.d_e     = TS.d_HII + TS.d_HeII / 4.0 + TS.d_HeIII / 2.0;
  e_dot_prev = fabs(TS.d_e - e_dot_prev) / fmax(dt, TINY_NUMBER);

  // Update internal energy
  TS.U += U_dot / TS.d * dt;
  #ifdef TEMPERATURE_FLOOR
  if (TS.get_temperature(Chem_H.gamma) < TEMP_FLOOR) TS.U = TS.compute_U(TEMP_FLOOR, Chem_H.gamma);
  #endif
  if (print > 1) printf("Updated U: %e \n", TS.U);
}

__global__ void Update_Chemistry_kernel(Real *dev_conserved, const Real *dev_rf, int nx, int ny, int nz, int n_ghost,
                                        int n_fields, Real dt_hydro, Chemistry_Header Chem_H)
{
  int id, xid, yid, zid, n_cells, n_iter;
  Real d, d_inv, vx, vy, vz;
  Real GE, E_kin, dt_chem, t_chem;
  Real current_a, a3, a2;


  // unit conversion
  Real density_conversion, energy_conversion;
  density_conversion = Chem_H.density_conversion;
  energy_conversion  = Chem_H.energy_conversion;

  //rates of change
  Real U_dot, HI_dot, e_dot, HI_dot_prev, e_dot_prev, temp_prev;
  Real k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI, k_coll_i_HI_HeI;
  Real k_recomb_HII, k_recomb_HeII, k_recomb_HeIII;
  float photo_i_HI, photo_i_HeI, photo_i_HeII;
  float photo_h_HI, photo_h_HeI, photo_h_HeII;
  Real correct_H, correct_He;

  n_cells = nx * ny * nz;

  // get a global thread ID
  id        = threadIdx.x + blockIdx.x * blockDim.x;
  zid       = id / (nx * ny);
  yid       = (id - zid * nx * ny) / nx;
  xid       = id - zid * nx * ny - yid * nx;
  int print = 0;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost - 1 && xid < nx - n_ghost && yid > n_ghost - 1 && yid < ny - n_ghost && zid > n_ghost - 1 &&
      zid < nz - n_ghost) {
    d     = dev_conserved[id];
    d_inv = 1.0 / d;
    vx    = dev_conserved[1 * n_cells + id] * d_inv;
    vy    = dev_conserved[2 * n_cells + id] * d_inv;
    vz    = dev_conserved[3 * n_cells + id] * d_inv;
    E_kin = 0.5 * d * (vx * vx + vy * vy + vz * vz);
  #ifdef DE
    GE = dev_conserved[(n_fields - 1) * n_cells + id];
  #else
    GE = dev_conserved[4 * n_cells + id] - E_kin;
  #endif

    if (xid == n_ghost && yid == n_ghost && zid == n_ghost) {
      /// print = 2;
    }

    // Convert to cgs units
    current_a = 1;
    a2 = 1;
    a3 = 1;

    //When doing cosmological integrations
    //factors of the scale factor are needed
    //to convert from the cosmological units
    //to proper units.
    //note that for density, we need factors
    //of a^2 and for density we need factors 
    //of a^3
  #ifdef COSMOLOGY
    current_a = 1 / (Chem_H.current_z + 1);
    a2        = current_a * current_a;      //a^2
    a3        = a2 * current_a;             //a^3
  #endif // COSMOLOGY

    d  *= density_conversion / a3;
    GE *= energy_conversion  / a2;

  #ifdef COSMOLOGY
    //convert cosmological hydro timestep into seconds
    //dt' = H0 dt/a^2
    //dt = a^2 dt' / H0
    //The chemistry H0 is in km/s/Mpc
    //So convert 1/H0 to kpc/(km/s) and then to seconds
    //by multiplying by the number of km in a kpc
    dt_hydro *= current_a * current_a / Chem_H.H0 * 1000 * KPC_KM;  //dt_hydro in proper seconds
  #endif  // COSMOLOGY


    // Initialize the thermal state
    Thermal_State TS;
    TS.d       = dev_conserved[id] / a3;
    TS.d_HI    = dev_conserved[5 * n_cells + id] / a3;
    TS.d_HII   = dev_conserved[6 * n_cells + id] / a3;
    TS.d_HeI   = dev_conserved[7 * n_cells + id] / a3;
    TS.d_HeII  = dev_conserved[8 * n_cells + id] / a3;
    TS.d_HeIII = dev_conserved[9 * n_cells + id] / a3;

    //set the thermal state internal
    //energy in km/s
    TS.U = GE * d_inv / (KM_CGS*KM_CGS);

    // Ceiling species
    TS.d_HI    = fmax(TS.d_HI, TINY_NUMBER);
    TS.d_HII   = fmax(TS.d_HII, TINY_NUMBER);
    TS.d_HeI   = fmax(TS.d_HeI, TINY_NUMBER);
    TS.d_HeII  = fmax(TS.d_HeII, TINY_NUMBER);
    TS.d_HeIII = fmax(TS.d_HeIII, 1e-5 * TINY_NUMBER);

    // Use charge conservation to determine electron fraction
    TS.d_e = TS.d_HII + TS.d_HeII / 4.0 + TS.d_HeIII / 2.0;

    // Compute temperature at first iteration
    temp_prev = TS.get_temperature(Chem_H.gamma);

    // Get the photoheating and photoionization rates at current epoch
    Get_Current_Photo_Rates(Chem_H, dev_rf, id, n_cells, photo_i_HI, photo_i_HeI, photo_i_HeII, photo_h_HI, photo_h_HeI,
                            photo_h_HeII, print);

    // Perform subcycling 
    HI_dot_prev = 0;
    e_dot_prev  = 0;
    n_iter      = 0;
    t_chem      = 0;
    while (t_chem < dt_hydro) {

      //BRANT: DOES THIS OVER RUN dt_hydro?  what prevents t_chem from exceeding dt_hydro?
      if (print != 0) printf("########################################## Iter %d \n", n_iter);

      U_dot = Get_Cooling_Rates(TS, Chem_H, Chem_H.dens_number_conv, Chem_H.current_z, temp_prev, photo_h_HI,
                                photo_h_HeI, photo_h_HeII, print);

      Get_Reaction_Rates(TS, Chem_H, k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI, k_coll_i_HI_HeI,
                         k_recomb_HII, k_recomb_HeII, k_recomb_HeIII, print);
      if (print > 1) {
        printf("k_photo_ion_HI: %e \n", photo_i_HI);
        printf("k_photo_ion_HeI: %e \n", photo_i_HeI);
        printf("k_photo_ion_HeII: %e \n", photo_i_HeII);
      }

      dt_chem =
          Get_Chemistry_dt(TS, Chem_H, HI_dot, e_dot, U_dot, k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI,
                           k_coll_i_HI_HeI, k_recomb_HII, k_recomb_HeII, k_recomb_HeIII, photo_i_HI, photo_i_HeI,
                           photo_i_HeII, n_iter, HI_dot_prev, e_dot_prev, t_chem, dt_hydro, print);

      Update_Step(TS, Chem_H, dt_chem, U_dot, k_coll_i_HI, k_coll_i_HeI, k_coll_i_HeII, k_coll_i_HI_HI, k_coll_i_HI_HeI,
                  k_recomb_HII, k_recomb_HeII, k_recomb_HeIII, photo_i_HI, photo_i_HeI, photo_i_HeII, HI_dot_prev,
                  e_dot_prev, temp_prev, print);

      t_chem += dt_chem;
      n_iter += 1;
      if (n_iter == Chem_H.max_iter) break;
    }
    if (print > 1) printf("Chem_GPU: N Iter:  %d\n", n_iter);

    // Make consistent abundances with the H and He density
    correct_H  = Chem_H.H_fraction * TS.d / (TS.d_HI + TS.d_HII);
    correct_He = (1.0 - Chem_H.H_fraction) * TS.d / (TS.d_HeI + TS.d_HeII + TS.d_HeIII);
    TS.d_HI    *= correct_H;
    TS.d_HII   *= correct_H;
    TS.d_HeI   *= correct_He;
    TS.d_HeII  *= correct_He;
    TS.d_HeIII *= correct_He;

    // Write the Updated Thermal State
    dev_conserved[5 * n_cells + id] = TS.d_HI    * a3;  //comoving number density
    dev_conserved[6 * n_cells + id] = TS.d_HII   * a3;  //comoving number density
    dev_conserved[7 * n_cells + id] = TS.d_HeI   * a3;  //comoving number density
    dev_conserved[8 * n_cells + id] = TS.d_HeII  * a3;  //comoving number density
    dev_conserved[9 * n_cells + id] = TS.d_HeIII * a3;  //comoving number density
    d                               = (d / density_conversion) * a3;
    GE                              = TS.U * (KM_CGS*KM_CGS) / d_inv / energy_conversion * a2 ;
    dev_conserved[4 * n_cells + id] = GE + E_kin;
  #ifdef DE
    dev_conserved[(n_fields - 1) * n_cells + id] = GE;
  #endif

    if (print != 0) printf("###########################################\n");
    if (print != 0) printf("Updated HI:  %e\n", TS.d_HI / TS.d);
    if (print != 0) printf("Updated HII:  %e\n", TS.d_HII / TS.d);
    if (print != 0) printf("Updated HeI:  %e\n", TS.d_HeI / TS.d);
    if (print != 0) printf("Updated HeII:  %e\n", TS.d_HeII / TS.d);
    if (print != 0) printf("Updated HeIII:  %e\n", TS.d_HeIII / TS.d);
    if (print != 0) printf("Updated GE:  %e\n", GE);
    if (print != 0) printf("Updated E:   %e\n", dev_conserved[4 * n_cells + id]);
  }
}

void Do_Chemistry_Update(Real *dev_conserved, const Real *dev_rf, int nx, int ny, int nz, int n_ghost, int n_fields,
                         Real dt, Chemistry_Header &Chem_H)
{
  float time;
  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  cudaEventRecord(start, 0);

  int ngrid = (nx * ny * nz - 1) / TPB_CHEM + 1;
  dim3 dim1dGrid(ngrid, 1, 1);
  dim3 dim1dBlock(TPB_CHEM, 1, 1);
  hipLaunchKernelGGL(Update_Chemistry_kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, dev_rf, nx, ny, nz, n_ghost,
                     n_fields, dt, Chem_H);

  CudaCheckError();
  cudaEventRecord(stop, 0);
  cudaEventSynchronize(stop);
  cudaEventElapsedTime(&time, start, stop);
  Chem_H.runtime_chemistry_step = (Real)time / 1000;  // (Convert ms to secs )
  cudaEventDestroy(start);
  cudaEventDestroy(stop);
}

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // Reaction and cooling rates from Grackle

// Calculation of k1 (HI + e --> HII + 2e)
// k1_rate
__host__ __device__ Real coll_i_HI_rate(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);

  Real k1 = exp(-32.71396786375 + 13.53655609057 * logT_ev - 5.739328757388 * pow(logT_ev, 2) +
                1.563154982022 * pow(logT_ev, 3) - 0.2877056004391 * pow(logT_ev, 4) +
                0.03482559773736999 * pow(logT_ev, 5) - 0.00263197617559 * pow(logT_ev, 6) +
                0.0001119543953861 * pow(logT_ev, 7) - 2.039149852002e-6 * pow(logT_ev, 8)) /
            units;
  if (T_ev <= 0.8) {
    k1 = fmax(TINY_NUMBER, k1);
  }
  return k1;
}

// Calculation of k3 (HeI + e --> HeII + 2e)
//  k3_rate
__host__ __device__ Real coll_i_HeI_rate(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);

  if (T_ev > 0.8) {
    return exp(-44.09864886561001 + 23.91596563469 * logT_ev - 10.75323019821 * pow(logT_ev, 2) +
               3.058038757198 * pow(logT_ev, 3) - 0.5685118909884001 * pow(logT_ev, 4) +
               0.06795391233790001 * pow(logT_ev, 5) - 0.005009056101857001 * pow(logT_ev, 6) +
               0.0002067236157507 * pow(logT_ev, 7) - 3.649161410833e-6 * pow(logT_ev, 8)) /
           units;
  } else {
    return TINY_NUMBER;
  }
}

// Calculation of k4 (HeII + e --> HeI + photon)
//  k4_rate
__host__ __device__ Real recomb_HeII_rate(Real T, Real units, bool use_case_B)
{
  Real T_ev    = T / 11605.0;
  // If case B recombination on.
  if (use_case_B) {
    return 1.26e-14 * pow(5.7067e5 / T, 0.75) / units;
  }

  // If case B recombination off.
  if (T_ev > 0.8) {
    return (1.54e-9 * (1.0 + 0.3 / exp(8.099328789667 / T_ev)) / (exp(40.49664394833662 / T_ev) * pow(T_ev, 1.5)) +
            3.92e-13 / pow(T_ev, 0.6353)) /
           units;
  } else {
    return 3.92e-13 / pow(T_ev, 0.6353) / units;
  }
}
// k4_rate Case A
__host__ __device__ Real recomb_HeII_rate_case_A(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  if (T_ev > 0.8) {
    return (1.54e-9 * (1.0 + 0.3 / exp(8.099328789667 / T_ev)) / (exp(40.49664394833662 / T_ev) * pow(T_ev, 1.5)) +
            3.92e-13 / pow(T_ev, 0.6353)) /
           units;
  } else {
    return 3.92e-13 / pow(T_ev, 0.6353) / units;
  }
}
// k4_rate Case B
__host__ __device__ Real recomb_HeII_rate_case_B(Real T, Real units)
{
  // If case B recombination on.
  return 1.26e-14 * pow(5.7067e5 / T, 0.75) / units;
}

// Calculation of k2 (HII + e --> HI + photon)
//  k2_rate
__host__ __device__ Real recomb_HII_rate(Real T, Real units, bool use_case_B)
{
  if (use_case_B) {
    if (T < 1.0e9) {
      return 4.881357e-6 * pow(T, -1.5) * pow((1.0 + 1.14813e2 * pow(T, -0.407)), -2.242) / units;
    } else {
      return TINY_NUMBER;
    }
  } else {
    if (T > 5500) {
      // Convert temperature to appropriate form.
      Real T_ev    = T / K_EV;
      Real logT_ev = log(T_ev);

      return exp(-28.61303380689232 - 0.7241125657826851 * logT_ev - 0.02026044731984691 * pow(logT_ev, 2) -
                 0.002380861877349834 * pow(logT_ev, 3) - 0.0003212605213188796 * pow(logT_ev, 4) -
                 0.00001421502914054107 * pow(logT_ev, 5) + 4.989108920299513e-6 * pow(logT_ev, 6) +
                 5.755614137575758e-7 * pow(logT_ev, 7) - 1.856767039775261e-8 * pow(logT_ev, 8) -
                 3.071135243196595e-9 * pow(logT_ev, 9)) /
             units;
    } else {
      return recomb_HeII_rate(T, units, use_case_B);
    }
  }
}
// k2_rate Case A
__host__ __device__ Real recomb_HII_rate_case_A(Real T, Real units)
{
  if (T > 5500) {
    // Convert temperature to appropriate form.
    Real T_ev    = T / K_EV;
    Real logT_ev = log(T_ev);

    return exp(-28.61303380689232 - 0.7241125657826851 * logT_ev - 0.02026044731984691 * pow(logT_ev, 2) -
               0.002380861877349834 * pow(logT_ev, 3) - 0.0003212605213188796 * pow(logT_ev, 4) -
               0.00001421502914054107 * pow(logT_ev, 5) + 4.989108920299513e-6 * pow(logT_ev, 6) +
               5.755614137575758e-7 * pow(logT_ev, 7) - 1.856767039775261e-8 * pow(logT_ev, 8) -
               3.071135243196595e-9 * pow(logT_ev, 9)) /
           units;
  } else {
    return recomb_HeII_rate_case_A(T, units);
  }
}

// k2_rate Case B
__host__ __device__ Real recomb_HII_rate_case_B(Real T, Real units)
{
  if (T < 1.0e9) {
    auto ret = 4.881357e-6 * pow(T, -1.5) * pow((1.0 + 1.14813e2 * pow(T, -0.407)), -2.242);
    return ret / units;
  } else {
    return TINY_NUMBER;
  }
}

__host__ __device__ Real recomb_HII_rate_case_Iliev1(Real T, Real units) { return 2.59e-13 / units; }

// Calculation of k5 (HeII + e --> HeIII + 2e)
//  k5_rate
__host__ __device__ Real coll_i_HeII_rate(Real T, Real units)
{
  Real T_ev    = T / 11605.0;
  Real logT_ev = log(T_ev);

  Real k5;
  if (T_ev > 0.8) {
    k5 = exp(-68.71040990212001 + 43.93347632635 * logT_ev - 18.48066993568 * pow(logT_ev, 2) +
             4.701626486759002 * pow(logT_ev, 3) - 0.7692466334492 * pow(logT_ev, 4) +
             0.08113042097303 * pow(logT_ev, 5) - 0.005324020628287001 * pow(logT_ev, 6) +
             0.0001975705312221 * pow(logT_ev, 7) - 3.165581065665e-6 * pow(logT_ev, 8)) /
         units;
  } else {
    k5 = TINY_NUMBER;
  }
  return k5;
}

// Calculation of k6 (HeIII + e --> HeII + photon)
//  k6_rate
__host__ __device__ Real recomb_HeIII_rate(Real T, Real units, bool use_case_B)
{
  Real k6;
  // Has case B recombination setting.
  if (use_case_B) {
    if (T < 1.0e9) {
      k6 = 7.8155e-5 * pow(T, -1.5) * pow((1.0 + 2.0189e2 * pow(T, -0.407)), -2.242) / units;
    } else {
      k6 = TINY_NUMBER;
    }
  } else {
    k6 = 3.36e-10 / sqrt(T) / pow(T / 1.0e3, 0.2) / (1.0 + pow(T / 1.0e6, 0.7)) / units;
  }
  return k6;
}
// k6_rate Case A
__host__ __device__ Real recomb_HeIII_rate_case_A(Real T, Real units)
{
  Real k6;
  // Has case B recombination setting.
  k6 = 3.36e-10 / sqrt(T) / pow(T / 1.0e3, 0.2) / (1.0 + pow(T / 1.0e6, 0.7)) / units;
  return k6;
}
// k6_rate Case B
__host__ __device__ Real recomb_HeIII_rate_case_B(Real T, Real units)
{
  Real k6;
  // Has case B recombination setting.
  if (T < 1.0e9) {
    k6 = 7.8155e-5 * pow(T, -1.5) * pow((1.0 + 2.0189e2 * pow(T, -0.407)), -2.242) / units;
  } else {
    k6 = TINY_NUMBER;
  }
  return k6;
}

// Calculation of k57 (HI + HI --> HII + HI + e)
//  k57_rate
__host__ __device__ Real coll_i_HI_HI_rate(Real T, Real units)
{
  // These rate coefficients are from Lenzuni, Chernoff & Salpeter (1991).
  // k57 value based on experimental cross-sections from Gealy & van Zyl (1987).
  if (T > 3.0e3) {
    return 1.2e-17 * pow(T, 1.2) * exp(-1.578e5 / T) / units;
  } else {
    return TINY_NUMBER;
  }
}

// Calculation of k58 (HI + HeI --> HII + HeI + e)
//  k58_rate
__host__ __device__ Real coll_i_HI_HeI_rate(Real T, Real units)
{
  // These rate coefficients are from Lenzuni, Chernoff & Salpeter (1991).
  // k58 value based on cross-sections from van Zyl, Le & Amme (1981).
  if (T > 3.0e3) {
    return 1.75e-17 * pow(T, 1.3) * exp(-1.578e5 / T) / units;
  } else {
    return TINY_NUMBER;
  }
}

// Calculation of ceHI.
//  Cooling collisional excitation HI
__host__ __device__ Real cool_ceHI_rate(Real T, Real units)
{
  return 7.5e-19 * exp(-fmin(log(HUGE_NUMBER), 118348.0 / T)) / (1.0 + sqrt(T / 1.0e5)) / units;
}

// Calculation of ceHeI.
//  Cooling collisional ionization HeI
__host__ __device__ Real cool_ceHeI_rate(Real T, Real units)
{
  return 9.1e-27 * exp(-fmin(log(HUGE_NUMBER), 13179.0 / T)) * pow(T, -0.1687) / (1.0 + sqrt(T / 1.0e5)) / units;
}

// Calculation of ceHeII.
//  Cooling collisional excitation HeII
__host__ __device__ Real cool_ceHeII_rate(Real T, Real units)
{
  return 5.54e-17 * exp(-fmin(log(HUGE_NUMBER), 473638.0 / T)) * pow(T, -0.3970) / (1.0 + sqrt(T / 1.0e5)) / units;
}

// Calculation of ciHeIS.
//  Cooling collisional ionization HeIS
__host__ __device__ Real cool_ciHeIS_rate(Real T, Real units)
{
  return 5.01e-27 * pow(T, -0.1687) / (1.0 + sqrt(T / 1.0e5)) * exp(-fmin(log(HUGE_NUMBER), 55338.0 / T)) / units;
}

// Calculation of ciHI.
//  Cooling collisional ionization HI
__host__ __device__ Real cool_ciHI_rate(Real T, Real units)
{
  // Collisional ionization. Polynomial fit from Tom Abel.
  return 2.18e-11 * coll_i_HI_rate(T, 1) / units;
}

// Calculation of ciHeI.
//  Cooling collisional ionization HeI
__host__ __device__ Real cool_ciHeI_rate(Real T, Real units)
{
  // Collisional ionization. Polynomial fit from Tom Abel.
  return 3.94e-11 * coll_i_HeI_rate(T, 1) / units;
}

// Calculation of ciHeII.
//  Cooling collisional ionization HeII
__host__ __device__ Real cool_ciHeII_rate(Real T, Real units)
{
  // Collisional ionization. Polynomial fit from Tom Abel.
  return 8.72e-11 * coll_i_HeII_rate(T, 1) / units;
}

// Calculation of reHII.
//  Cooling recombination HII
__host__ __device__ Real cool_reHII_rate(Real T, Real units, bool use_case_B)
{
  Real lambdaHI = 2.0 * 157807.0 / T;
  if (use_case_B) {
    return 3.435e-30 * T * pow(lambdaHI, 1.970) / pow(1.0 + pow(lambdaHI / 2.25, 0.376), 3.720) / units;
  } else {
    return 1.778e-29 * T * pow(lambdaHI, 1.965) / pow(1.0 + pow(lambdaHI / 0.541, 0.502), 2.697) / units;
  }
}

// Calculation of reHII.
//  Cooling recombination HII Case A
__host__ __device__ Real cool_reHII_rate_case_A(Real T, Real units)
{
  Real lambdaHI = 2.0 * 157807.0 / T;
  return 1.778e-29 * T * pow(lambdaHI, 1.965) / pow(1.0 + pow(lambdaHI / 0.541, 0.502), 2.697) / units;
}

// Calculation of reHII.
//  Cooling recombination HII Case B
__host__ __device__ Real cool_reHII_rate_case_B(Real T, Real units)
{
  Real lambdaHI = 2.0 * 157807.0 / T;
  return 3.435e-30 * T * pow(lambdaHI, 1.970) / pow(1.0 + pow(lambdaHI / 2.25, 0.376), 3.720) / units;
}

// Calculation of reHII.
//  Cooling recombination HeII
__host__ __device__ Real cool_reHeII1_rate(Real T, Real units, bool use_case_B)
{
  Real lambdaHeII = 2.0 * 285335.0 / T;
  if (use_case_B) {
    return 1.26e-14 * KB * T * pow(lambdaHeII, 0.75) / units;
  } else {
    return 3e-14 * KB * T * pow(lambdaHeII, 0.654) / units;
  }
}

// Calculation of reHII.
//  Cooling recombination HeII Case A
__host__ __device__ Real cool_reHeII1_rate_case_A(Real T, Real units)
{
  Real lambdaHeII = 2.0 * 285335.0 / T;
  return 3e-14 * KB * T * pow(lambdaHeII, 0.654) / units;
}

// Calculation of reHII.
//  Cooling recombination HeII Case B
__host__ __device__ Real cool_reHeII1_rate_case_B(Real T, Real units)
{
  Real lambdaHeII = 2.0 * 285335.0 / T;
  return 1.26e-14 * KB * T * pow(lambdaHeII, 0.75) / units;
}

// Calculation of reHII2.
//  Cooling recombination HeII Dielectronic
__host__ __device__ Real cool_reHeII2_rate(Real T, Real units)
{
  // Dielectronic recombination (Cen, 1992).
  return 1.24e-13 * pow(T, -1.5) * exp(-fmin(log(HUGE_NUMBER), 470000.0 / T)) *
         (1.0 + 0.3 * exp(-fmin(log(HUGE_NUMBER), 94000.0 / T))) / units;
}

// Calculation of reHIII.
//  Cooling recombination HeIII
__host__ __device__ Real cool_reHeIII_rate(Real T, Real units, bool use_case_B)
{
  Real lambdaHeIII = 2.0 * 631515.0 / T;
  if (use_case_B) {
    return 8.0 * 3.435e-30 * T * pow(lambdaHeIII, 1.970) / pow(1.0 + pow(lambdaHeIII / 2.25, 0.376), 3.720) / units;
  } else {
    return 8.0 * 1.778e-29 * T * pow(lambdaHeIII, 1.965) / pow(1.0 + pow(lambdaHeIII / 0.541, 0.502), 2.697) / units;
  }
}

// Calculation of reHIII.
//  Cooling recombination HeIII Case A
__host__ __device__ Real cool_reHeIII_rate_case_A(Real T, Real units)
{
  Real lambdaHeIII = 2.0 * 631515.0 / T;
  return 8.0 * 1.778e-29 * T * pow(lambdaHeIII, 1.965) / pow(1.0 + pow(lambdaHeIII / 0.541, 0.502), 2.697) / units;
}

// Calculation of reHIII.
//  Cooling recombination HeIII Case B
__host__ __device__ Real cool_reHeIII_rate_case_B(Real T, Real units)
{
  Real lambdaHeIII = 2.0 * 631515.0 / T;
  return 8.0 * 3.435e-30 * T * pow(lambdaHeIII, 1.970) / pow(1.0 + pow(lambdaHeIII / 2.25, 0.376), 3.720) / units;
}
// Calculation of brem.
//  Cooling Bremsstrahlung
__host__ __device__ Real cool_brem_rate(Real T, Real units)
{
  return 1.43e-27 * sqrt(T) * (1.1 + 0.34 * exp(-pow(5.5 - log10(T), 2) / 3.0)) / units;
}

#endif
